#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <cstdio>

static void HandleError(hipError_t err, const char*file, int line)
{
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
                exit(EXIT_FAILURE);
        }
}

#define CUDA_HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void kernel_addbsquared(const double* a, const double* b, int size, double* c)
{
        const int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i < size)
        {
                c[i] = a[i] + b[i] * b[i];
        }
}

namespace ncv
{
        int cuda::count_devices()
        {
                int count = 0;
                CUDA_HANDLE_ERROR(hipGetDeviceCount(&count));

                return count;
        }

        hipDeviceProp_t cuda::get_device_properties(int device)
        {
                hipDeviceProp_t prop;
                CUDA_HANDLE_ERROR(hipGetDeviceProperties(&prop, device));

                return prop;
        }

        bool cuda::print_info()
        {
                const int count = cuda::count_devices();
                for (int i = 0; i < count; i ++)
                {
                        const hipDeviceProp_t prop = get_device_properties(i);

                        printf("CUDA device [%d/%d]: name = %s\n", i + 1, count, prop.name);
                        printf("CUDA device [%d/%d]: compute capability = %d.%d\n", i + 1, count, prop.major, prop.minor);
                        printf("CUDA device [%d/%d]: clock rate = %d\n", i + 1, count, prop.clockRate);
                        printf("CUDA device [%d/%d]: global mem = %ld\n", i + 1, count, prop.totalGlobalMem);
                        printf("CUDA device [%d/%d]: constant Mem = %ld\n", i + 1, count, prop.totalConstMem);
                        printf("CUDA device [%d/%d]: mem pitch = %ld\n", i + 1, count, prop.memPitch);
                        printf("CUDA device [%d/%d]: texture alignment = %ld\n", i + 1, count, prop.textureAlignment);
                        printf("CUDA device [%d/%d]: multiprocessor count = %d\n", i + 1, count, prop.multiProcessorCount);
                        printf("CUDA device [%d/%d]: shared mem per mp = %ld\n", i + 1, count, prop.sharedMemPerBlock);
                        printf("CUDA device [%d/%d]: registers per mp = %d\n", i + 1, count, prop.regsPerBlock);
                        printf("CUDA device [%d/%d]: threads in warp = %d\n", i + 1, count, prop.warpSize);
                        printf("CUDA device [%d/%d]: max threads per block = %d\n", i + 1, count, prop.maxThreadsPerBlock);
                        printf("CUDA device [%d/%d]: max thread dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
                        printf("CUDA device [%d/%d]: max grid dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
                        printf("\n");
                }

                return true;
        }

        dim3 make_size(int size, int device = 0)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3((size + prop.maxThreadsPerBlock - 1) / prop.maxThreadsPerBlock, 1, 1);
        }

        dim3 make_block_size(int size, int device = 0)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3(prop.maxThreadsPerBlock, 1, 1);
        }

        bool cuda::addbsquared(const vector_t<double>& a, const vector_t<double>& b, vector_t<double>& c)
        {
                if (    a.size() != c.size() ||
                        b.size() != c.size())
                {
                        return false;
                }

                else
                {
                        const dim3 ksize = make_size(a.size());
                        const dim3 bsize = make_block_size(a.size());

                        kernel_addbsquared<<<ksize, bsize>>>(a.data(), b.data(), a.size(), c.data());

                        return hipGetLastError() == hipSuccess;
                }
        }
}
