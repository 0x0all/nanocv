#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>

namespace ncv
{
        bool cuda::copyToDevice(const double* h_data, int size)
        {
                thrust::device_vector<double> d_data(size);
                thrust::copy(h_data, h_data + size, d_data.begin());

                return true;
        }

        bool cuda::copyFromDevice(const double* d_data, int size, double* h_data)
        {
                return false;
        }
}
