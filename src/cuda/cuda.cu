#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <cstdio>

static void HandleError(hipError_t err, const char*file, int line)
{
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
                exit(EXIT_FAILURE);
        }
}

#define CUDA_HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void kernel_addbsquared(const double* a, const double* b, int size, double* c)
{
        const int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i < size)
        {
                c[i] = a[i] + b[i] * b[i];
        }
}

namespace ncv
{
        int cuda::count_devices()
        {
                int count = 0;
                CUDA_HANDLE_ERROR(hipGetDeviceCount(&count));

                return count;
        }

        hipDeviceProp_t cuda::get_device_properties(int device)
        {
                hipDeviceProp_t prop;
                CUDA_HANDLE_ERROR(hipGetDeviceProperties(&prop, device));

                return prop;
        }

        bool cuda::print_info()
        {
                const int count = cuda::count_devices();
                for (int i = 0; i < count; i ++)
                {
                        const hipDeviceProp_t prop = get_device_properties(i);

                        printf("CUDA device [%d/%d]: name = %s\n", i + 1, count, prop.name);
                        printf("CUDA device [%d/%d]: compute capability = %d.%d\n", i + 1, count, prop.major, prop.minor);
                        printf("CUDA device [%d/%d]: clock rate = %d\n", i + 1, count, prop.clockRate);
                        printf("CUDA device [%d/%d]: global mem = %ld\n", i + 1, count, prop.totalGlobalMem);
                        printf("CUDA device [%d/%d]: constant Mem = %ld\n", i + 1, count, prop.totalConstMem);
                        printf("CUDA device [%d/%d]: mem pitch = %ld\n", i + 1, count, prop.memPitch);
                        printf("CUDA device [%d/%d]: texture alignment = %ld\n", i + 1, count, prop.textureAlignment);
                        printf("CUDA device [%d/%d]: multiprocessor count = %d\n", i + 1, count, prop.multiProcessorCount);
                        printf("CUDA device [%d/%d]: shared mem per mp = %ld\n", i + 1, count, prop.sharedMemPerBlock);
                        printf("CUDA device [%d/%d]: registers per mp = %d\n", i + 1, count, prop.regsPerBlock);
                        printf("CUDA device [%d/%d]: threads in warp = %d\n", i + 1, count, prop.warpSize);
                        printf("CUDA device [%d/%d]: max threads per block = %d\n", i + 1, count, prop.maxThreadsPerBlock);
                        printf("CUDA device [%d/%d]: max thread dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
                        printf("CUDA device [%d/%d]: max grid dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
                        printf("\n");
                }

                return true;
        }

        dim3 make_size(int size, int device = 0)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3((size + prop.maxThreadsPerBlock - 1) / prop.maxThreadsPerBlock, 1, 1);
        }

        dim3 make_block_size(int size, int device = 0)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3(prop.maxThreadsPerBlock, 1, 1);
        }

        namespace cuda
        {
                struct device_buffer_impl_t
                {
                        thrust::device_vector<double>   m_data;
                };
        }

        cuda::device_buffer_t::device_buffer_t(int size)
                :       m_impl(new cuda::device_buffer_impl_t)
        {
                m_impl->m_data.resize(size);
        }

        cuda::device_buffer_t::~device_buffer_t()
        {
                delete m_impl;
        }

        int cuda::device_buffer_t::size() const
        {
                return static_cast<int>(m_impl->m_data.size());
        }

        bool cuda::device_buffer_t::empty() const
        {
                return m_impl->m_data.empty();
        }

        const cuda::device_buffer_impl_t& cuda::device_buffer_t::get() const
        {
                return *m_impl;
        }

        cuda::device_buffer_impl_t& cuda::device_buffer_t::get()
        {
                return *m_impl;
        }

        bool cuda::device_buffer_t::copyToDevice(const double* h_data) const
        {
                thrust::copy(h_data, h_data + size(), m_impl->m_data.begin());
                return true;
        }

        bool cuda::device_buffer_t::copyFromDevice(double* h_data) const
        {
                thrust::copy(m_impl->m_data.begin(), m_impl->m_data.end(), h_data);
                return false;
        }

        bool cuda::addbsquared(const device_buffer_t& a, const device_buffer_t& b, device_buffer_t& c)
        {
                if (    a.size() != c.size() ||
                        b.size() != c.size())
                {
                        return false;
                }

                else
                {
                        const thrust::device_vector<double>& d_a = a.get().m_data;
                        const thrust::device_vector<double>& d_b = b.get().m_data;
                        thrust::device_vector<double>& d_c = c.get().m_data;

                        const dim3 ksize = make_size(a.size());
                        const dim3 bsize = make_block_size(a.size());

                        kernel_addbsquared<<<ksize, bsize>>>(
                                thrust::raw_pointer_cast(&d_a[0]),
                                thrust::raw_pointer_cast(&d_b[0]),
                                d_a.size(),
                                thrust::raw_pointer_cast(&d_c[0]));

                        return true;
                }
        }
}
