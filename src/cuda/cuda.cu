#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <cstdio>

static void HandleError(hipError_t err, const char*file, int line)
{
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
                exit(EXIT_FAILURE);
        }
}

#define CUDA_HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

namespace ncv
{
        int cuda::count_devices()
        {
                int count = 0;
                CUDA_HANDLE_ERROR(hipGetDeviceCount(&count));

                return count;
        }

        hipDeviceProp_t cuda::get_device_properties(int device)
        {
                hipDeviceProp_t prop;
                CUDA_HANDLE_ERROR(hipGetDeviceProperties(&prop, device));

                return prop;
        }

        bool cuda::print_info()
        {
                const int count = cuda::count_devices();
                for (int i = 0; i < count; i ++)
                {
                        const hipDeviceProp_t prop = get_device_properties(i);

                        printf("CUDA device [%d/%d]: name = %s\n", i + 1, count, prop.name);
                        printf("CUDA device [%d/%d]: compute capability = %d.%d\n", i + 1, count, prop.major, prop.minor);
                        printf("CUDA device [%d/%d]: clock rate = %d\n", i + 1, count, prop.clockRate);
                        printf("CUDA device [%d/%d]: global mem = %ld\n", i + 1, count, prop.totalGlobalMem);
                        printf("CUDA device [%d/%d]: constant Mem = %ld\n", i + 1, count, prop.totalConstMem);
                        printf("CUDA device [%d/%d]: mem pitch = %ld\n", i + 1, count, prop.memPitch);
                        printf("CUDA device [%d/%d]: texture alignment = %ld\n", i + 1, count, prop.textureAlignment);
                        printf("CUDA device [%d/%d]: multiprocessor count = %d\n", i + 1, count, prop.multiProcessorCount);
                        printf("CUDA device [%d/%d]: shared mem per mp = %ld\n", i + 1, count, prop.sharedMemPerBlock);
                        printf("CUDA device [%d/%d]: registers per mp = %d\n", i + 1, count, prop.regsPerBlock);
                        printf("CUDA device [%d/%d]: threads in warp = %d\n", i + 1, count, prop.warpSize);
                        printf("CUDA device [%d/%d]: max threads per block = %d\n", i + 1, count, prop.maxThreadsPerBlock);
                        printf("CUDA device [%d/%d]: max thread dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
                        printf("CUDA device [%d/%d]: max grid dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
                        printf("\n");
                }

                return true;
        }

        dim3 cuda::make_block1d_count(int size, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3((size + prop.maxThreadsPerBlock - 1) / prop.maxThreadsPerBlock,
                            1,
                            1);
        }

        dim3 cuda::make_block2d_count(int rows, int cols, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3((cols + prop.maxThreadsPerBlock - 1) / prop.maxThreadsPerBlock,
                            (rows + prop.maxThreadsPerBlock - 1) / prop.maxThreadsPerBlock,
                            1);
        }

        dim3 cuda::make_block1d_size(int, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3(prop.maxThreadsPerBlock,
                            1,
                            1);
        }

        dim3 cuda::make_block2d_size(int, int, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                return dim3(sqrt(prop.maxThreadsPerBlock),
                            sqrt(prop.maxThreadsPerBlock),
                            1);
        }
}
