#include "hip/hip_runtime.h"
#include "conv2d.h"
#include "hip/hip_runtime.h"

__global__ void kernel_conv2d(
        const ncv::cuda::matrix_t<double>& idata,
        const ncv::cuda::matrix_t<double>& kdata,
        ncv::cuda::matrix_t<double>& odata)
{
        const int c = threadIdx.x + blockIdx.x * blockDim.x;
        const int r = threadIdx.y + blockIdx.y * blockDim.y;
        
        const int orows = odata.rows();
        const int ocols = odata.cols();

        if (r < orows && c < ocols)
        {
                const int krows = kdata.rows();
                const int kcols = kdata.cols();

                double sum = 0;
                for (int kr = 0; kr < krows; kr ++)
                {
                        for (int kc = 0; kc < kcols; kc ++)
                        {
                                sum += idata(r + kr, c + kc) * kdata(kr, kc);
                        }
                }

                odata(r, c) = sum;
        }
}

__global__ void kernel_iconv2d(
        const ncv::cuda::matrix_t<double>& odata,
        const ncv::cuda::matrix_t<double>& kdata,
        ncv::cuda::matrix_t<double>& idata)
{
        const int c = threadIdx.x + blockIdx.x * blockDim.x;
        const int r = threadIdx.y + blockIdx.y * blockDim.y;
        
        const int irows = idata.rows();
        const int icols = idata.cols();

        if (r < irows && c < icols)
        {
                const int krows = kdata.rows();
                const int kcols = kdata.cols();
                
                const int orows = irows - krows + 1;
                const int ocols = icols - kcols + 1;

                const int krmin = max(0,     r - orows + 1);
                const int krmax = min(krows, r + 1);

                const int kcmin = max(0,     c - ocols + 1);
                const int kcmax = min(kcols, c + 1);

                double sum = 0;
                for (int kr = krmin; kr < krmax; kr ++)
                {
                        for (int kc = kcmin; kc < kcmax; kc ++)
                        {
                                sum += odata(r - kr, c - kc) * kdata(kr, kc);
                        }
                }

                idata(r, c) = sum;
        }
}

namespace ncv
{
        bool cuda::conv2d(
                const matrix_t<double>& idata, const matrix_t<double>& kdata, matrix_t<double>& odata,
                int device)
        {
                if (    odata.rows() + kdata.rows() != idata.rows() + 1 ||
                        odata.cols() + kdata.cols() != idata.cols() + 1)
                {
                        return false;
                }

                else
                {
                        const dim3 ksize = cuda::make_block2d_count(odata.rows(), odata.cols(), device);
                        const dim3 bsize = cuda::make_block2d_size(odata.rows(), odata.cols(), device);

                        kernel_conv2d<<<ksize, bsize>>>(idata, kdata, odata);

                        return hipGetLastError() == hipSuccess;
                }
        }

        bool cuda::iconv2d(
                const matrix_t<double>& odata, const matrix_t<double>& kdata, matrix_t<double>& idata,
                int device)
        {
                if (    odata.rows() + kdata.rows() != idata.rows() + 1 ||
                        odata.cols() + kdata.cols() != idata.cols() + 1)
                {
                        return false;
                }

                else
                {
                        const dim3 ksize = cuda::make_block2d_count(idata.rows(), idata.cols(), device);
                        const dim3 bsize = cuda::make_block2d_size(idata.rows(), idata.cols(), device);

                        kernel_iconv2d<<<ksize, bsize>>>(odata, kdata, idata);

                        return hipGetLastError() == hipSuccess;
                }
        }
}
